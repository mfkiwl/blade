
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

template<size_t N>
__global__ void checker_complex(const hipFloatComplex* a, const hipFloatComplex* b,
        unsigned long long int* result) {
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadID; i < N; i += numThreads) {
#ifdef DEBUG
        if (threadIdx.x == 0 && blockIdx.x == 0) {
            for (int i = 0; i < N; i++) {
                const char* status = (cuCabsf(cuCsubf(a[i], b[i])) > 0.01) ? "NOK" : " OK";
                printf("[%04d] %s %+f%+fj != %+f%+fj\n", i, status, a[i].x, a[i].y, b[i].x, b[i].y);
            }
        }
#endif
        if (hipCabsf(hipCsubf(a[i], b[i])) > 0.01) {
            atomicAdd(result, 1);
        }
    }
}

template<typename T, size_t N>
__global__ void checker(const T* a, const T* b, unsigned long long int* result) {
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadID; i < N; i += numThreads) {
#ifdef DEBUG
        if (threadIdx.x == 0 && blockIdx.x == 0) {
            for (int i = 0; i < N; i++) {
                const char* status = (abs(a[i], b[i]) > 0.01) ? "NOK" : " OK";
                printf("[%04d] %s %f != %f\n", i, status, (T)a[i], (T)b[i]);
            }
        }
#endif
        if (abs(a[i] - b[i]) > 0.01) {
            atomicAdd(result, 1);
        }
    }
}
